#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void matrixmul(int *a, int *b, int *c, int c1, int c2)
{
	int row_id = threadIdx.x; 
	for(int i = 0; i < c2; i++)
	{
		for(int j = 0; j < c1; j++)
			c[row_id*c2 + i] = c[row_id*c2 + i] + a[row_id*c1 + j]*b[j*c2 + c2];
	}
}

int main()
{
	int r1, c1, c2; 
	printf("Enter row dim of A: ");
	scanf("%d", &r1); 
	printf("Enter col dim of A/row dim of B: "); 
	scanf("%d", &c1); 
	printf("Enter col dim of second matrix: "); 
	scanf("%d", &c2); 
	int a[r1][c1], b[c1][c2], c[r1][c2]; 
	printf("Enter elements in A: ");
	for(int i = 0; i < r1; i++)
		for(int j = 0; j < c1; j++)
			scanf("%d", &a[i][j]);  
	printf("Enter elements in B: "); 
	for(int i = 0; i < c1; i++)
		for(int j = 0; j < c2; j++)
			scanf("%d", &b[i][j]); 
	int *d_a, *d_b, *d_c; 
	hipMalloc((void**)&d_a, sizeof(int)*r1*c1);
	hipMalloc((void**)&d_b, sizeof(int)*c1*c2);
	hipMalloc((void**)&d_c, sizeof(int)*r1*c2); 
 	hipMemcpy(d_a, a, sizeof(int)*r1*c1, hipMemcpyHostToDevice);
 	hipMemcpy(d_b, b, sizeof(int)*c1*c2, hipMemcpyHostToDevice);
 	matrixmul<<<1, c1>>>(d_a, d_b, d_c, c1, c2); 
 	hipMemcpy(c, d_c, sizeof(int)*r1*c2, hipMemcpyDeviceToHost);
 	printf("The modified matrix is:\n"); 
	for(int i = 0; i < r1; i++)
		{
			for(int j = 0; j < c2; j++)
				printf("%d\t", c[i][j]); 
			printf("\n"); 
		}  

}