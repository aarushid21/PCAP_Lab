#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void rgbToGray(int *r, int *b, int *g, int * gray, int col_size)
{
	int id = threadIdx.x; 
	gray[id] = (r[id] + g[id] + b[id])/3; 
}


int main()
{
	int x, y;
	printf("Enter x dimension: ");
	scanf("%d", &x); 
	printf("Enter y dimension: "); 
	scanf("%d", &y); 
	int r_img[x][y], b_img[x][y], g_img[x][y], gray[x][y];
	printf("Enter image matrix for R channel: ");
	for(int i = 0; i < x; i++)
		for(int j = 0; j < y; j++)
			scanf("%d", &r_img[i][j]);  
	printf("Enter image matrix for G channel: ");
	for(int i = 0; i < x; i++)
		for(int j = 0; j < y; j++)
			scanf("%d", &g_img[i][j]); 
	printf("Enter image matrix for B channel: ");
	for(int i = 0; i < x; i++)
		for(int j = 0; j < y; j++)
			scanf("%d", &b_img[i][j]);   
	int *d_r_img, *d_b_img, *d_g_img, *d_gray; 
	int size = x*y*sizeof(int); 
	hipMalloc((void**)&d_r_img, size);
	hipMalloc((void**)&d_b_img, size);
	hipMalloc((void**)&d_g_img, size);
	hipMalloc((void**)&d_gray, size); 
	hipMemcpy(d_r_img, r_img, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b_img, b_img, size, hipMemcpyHostToDevice);
	hipMemcpy(d_g_img, g_img, size, hipMemcpyHostToDevice);
	rgbToGray<<<1, x*y>>>(d_r_img, d_b_img, d_g_img, d_gray, y);
	hipMemcpy(gray, d_gray, size, hipMemcpyDeviceToHost); 
	printf("The grayscaled version of the image is:\n"); 
	for(int i = 0; i < x; i++)
		{
			for(int j = 0; j < y; j++)
				printf("%d\t", gray[i][j]); 
			printf("\n"); 
		}
}

