#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void matrixmul(int *a, int *b, int *c, int r1, int c1, int c2)
{
	int c_id = threadIdx.x%r1; 
	for(int i = 0; i < r1; i++)
	{
		for(int j = 0; j < c1; j++)
			c[i*c2 + c_id] = c[i*c2 + c_id] + a[i*c1 + j]*b[j*c2 + i];
	}
}

int main()
{
	int r1, c1, c2; 
	printf("Enter row dim of A: ");
	scanf("%d", &r1); 
	printf("Enter col dim of A/row dim of B: "); 
	scanf("%d", &c1); 
	printf("Enter col dim of second matrix: "); 
	scanf("%d", &c2); 
	int a[r1][c1], b[c1][c2], c[r1][c2]; 
	printf("Enter elements in A: "); 
	for(int i = 0; i < r1; i++)
		for(int j = 0; j < c1; j++)
			scanf("%d", &a[i][j]);  
	printf("Enter elements in B: "); 
	for(int i = 0; i < c1; i++)
		for(int j = 0; j < c2; j++)
			scanf("%d", &b[i][j]); 
	int *d_a, *d_b, *d_c; 
	hipMalloc((void**)&d_a, sizeof(int)*r1*c1);
	hipMalloc((void**)&d_b, sizeof(int)*c1*c2);
	hipMalloc((void**)&d_c, sizeof(int)*r1*c2); 
 	hipMemcpy(d_a, a, sizeof(int)*r1*c1, hipMemcpyHostToDevice);
 	hipMemcpy(d_b, b, sizeof(int)*c1*c2, hipMemcpyHostToDevice);
 	matrixmul<<<1, c2>>>(d_a, d_b, d_c, r1, c1, c2); 
 	hipMemcpy(c, d_c, sizeof(int)*r1*c2, hipMemcpyDeviceToHost);
 	printf("The modified matrix is:\n"); 
	for(int i = 0; i < r1; i++)
		{
			for(int j = 0; j < c2; j++)
				printf("%d\t", c[i][j]); 
			printf("\n"); 
		}  

}