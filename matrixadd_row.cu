#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int*c, int cols)
{
	int id = threadIdx.x; 
	for(int i = id*cols; i < id*cols + cols; i++)
		c[i] = a[i] + b[i]; 
}

int main()
{
	int x, y; 
	printf("Enter number of rows: "); 
	scanf("%d", &x); 
	printf("Enter the number of columns: "); 
	scanf("%d", &y); 
	int a[x][y], b[x][y], c[x][y]; 
	printf("Enter elements in a: "); 
	for(int i = 0; i < x; i++)
		for(int j = 0; j < y; j++)
			scanf("%d", &a[i][j]);
	printf("Enter elements in b: "); 
	for(int i = 0; i < x; i++)
		for(int j = 0; j < y; j++)
			scanf("%d", &b[i][j]);
	int *d_a, *d_b, *d_c; 
	int size = x*y*sizeof(int); 
	hipMalloc((void**)&d_a, size); 
	hipMalloc((void**)&d_b, size); 
	hipMalloc((void**)&d_c, size); 
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice); 
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice); 
	add<<<1, x>>>(d_a, d_b, d_c, y);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost); 
	printf("The modified matrix is:\n"); 
	for(int i = 0; i < x; i++)
		{
			for(int j = 0; j < y; j++)
				printf("%d\t", c[i][j]); 
			printf("\n"); 
		}  
}