#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void matrixModify(int *a, int *b, int col_size)
{
	int id = threadIdx.x; 
	if(id%2 == 0)
	{
		int replace = 0; 
		for(int i = 0; i < blockDim.x; i++)
		{
			if(i%col_size == id%col_size)
				replace = replace + a[i]; 
		}
		b[id] = replace; 
	}
	else 
	{
		int row = id/col_size; 
		int replace = 0; 
		for(int i = row*col_size; i <row*col_size + col_size; i++)
			replace = replace + a[i]; 
		b[id] = replace; 
	}
}

int main()
{
	int x, y; 
	printf("Enter number of rows: "); 
	scanf("%d", &x); 
	printf("Enter the number of columns: "); 
	scanf("%d", &y); 
	int a[x][y], b[x][y]; 
	printf("Enter elements in a: "); 
	for(int i = 0; i < x; i++)
		for(int j = 0; j < y; j++)
			scanf("%d", &a[i][j]);
	int *d_a, *d_b; 
	int size = x*y*sizeof(int); 
	hipMalloc((void**)&d_a, size); 
	hipMalloc((void**)&d_b, size); 
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice); 
	matrixModify<<<1, x*y>>>(d_a, d_b, y);
	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost); 
	printf("The modified matrix is:\n"); 
	for(int i = 0; i < x; i++)
		{
			for(int j = 0; j < y; j++)
				printf("%d\t", b[i][j]); 
			printf("\n"); 
		}  
}