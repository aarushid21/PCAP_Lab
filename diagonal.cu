#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void diagonalModify(int *a, int col_size)
{
	int id = threadIdx.x; 
	int c_id = id%col_size; 
	int r_id = id/col_size; 
	if(c_id == r_id)
		a[id] = 0; 
	else if(r_id < c_id)
	{
		int fac = 1; 
		for(int i = 2; i <= a[id]; i++)
			fac = fac*i; 
		a[id] = fac; 
	}
	else if(r_id > c_id)
	{
		int sum = 0; 
		int temp = a[id]; 
		while(temp!= 0)
		{
			sum = sum + temp%10; 
			temp = temp/10;
		}
		a[id] = sum; 
	}
}

int main()
{
	int x, y; 
	printf("Enter number of rows: "); 
	scanf("%d", &x); 
	printf("Enter the number of columns: "); 
	scanf("%d", &y); 
	int a[x][y]; 
	printf("Enter elements in a: "); 
	for(int i = 0; i < x; i++)
		for(int j = 0; j < y; j++)
			scanf("%d", &a[i][j]);
	int *d_a; 
	int size = x*y*sizeof(int); 
	hipMalloc((void**)&d_a, size); 
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice); 
	diagonalModify<<<1, x*y>>>(d_a, y);
	hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost); 
	printf("The modified matrix is:\n"); 
	for(int i = 0; i < x; i++)
		{
			for(int j = 0; j < y; j++)
				printf("%d\t", b[i][j]); 
			printf("\n"); 
		}  
}